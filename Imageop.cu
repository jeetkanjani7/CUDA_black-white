#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "opencv2/imgproc/imgproc.hpp"

using namespace std;
using namespace cv;
	
__global__ void rgba_to_greyscale(uchar3 *rgbaImage, unsigned char* d_grayscale, int numRows, int numCols)
{
	
    //printf("%u",rgbaImage[threadIdx.x]);
    int absolute_position_x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int absolute_position_y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int Id = absolute_position_x + absolute_position_y*numCols;
    

    if ( absolute_position_x >= numCols ||  absolute_position_y >= numRows )
    { 
     return;
    }

    uchar3 rgba = rgbaImage[Id];

    float channelSum = 0.299f * rgba.x + 0.587f * rgba.y + 0.114f * rgba.z;
    
    d_grayscale[Id] = channelSum;
  
}

int main()
{

    Mat input,input_gray;   
    input = imread("/home/jeetkanjani7/scene.jpg",-1); 
    cout<<"total: "<<input.total();
    
    uchar3 *d_inputImage = new uchar3[input.total()];
    
    unsigned char *d_grayscale;
    unsigned char *h_grayscale = (unsigned char*)malloc(input.total());

    hipMalloc((void**)&d_grayscale,input.total());
  

   hipMalloc((void**)&d_inputImage,input.total()*sizeof(uchar3));
    hipMemcpy(d_inputImage, input.data,input.total()*sizeof(uchar3), hipMemcpyHostToDevice);
    
    int numrows = input.rows;
    int numcols = input.cols; 
    
    const dim3 blockSize(16, 16, 1);  //TODO
    const dim3 gridSize((numcols/blockSize.x),(numrows/blockSize.y),1);  //TODO
    
    cout<<"\ngridSize::"<<gridSize.x<<" "<<gridSize.y;
    cout<<"\nBlockSize::"<<blockSize.x<<" "<<blockSize.y<<endl;

   
    
    rgba_to_greyscale<<<gridSize, blockSize>>>(d_inputImage, d_grayscale, numrows, numcols);
    hipDeviceSynchronize();
    
   hipMemcpy(h_grayscale, d_grayscale,input.total(), hipMemcpyDeviceToHost);
     cout<<static_cast<int>(h_grayscale[7])<<endl;
    Mat img(numrows,numcols,CV_8UC1,h_grayscale,cv::Mat::AUTO_STEP);
    imshow("rgb",img); 
    cout<<"image dims: "<<img.size();
    hipDeviceSynchronize();
    
    printf("That's all!\n");
    waitKey(0);
     
    return 0;

}